#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
//#include <hip/device_functions.h>
//#define _USE_MATH_DEFINES
//#include <math.h>
#include "vis_sim.h"

texture<float, 3, hipReadModeNormalizedFloat> tex;

__global__ void find_vis( float *baseline, float *src_dir, float *src_int, float *src_index, float *freqs, float* mfreqs, int *N_fq_p, int *N_src_p, float *vis_arr, float *beam_arr, float *lmin, float *lmax, float *mmin, float *mmax, float *beamfqmin, float *beamfqmax) {
	//Inputs: Baseline is length 3 vector in nanoseconds, src_dir is N_src*3 array, src_int is an N_src array, src_index is a N_src array, freqs is an N_fq array of frequencies in GHz, mfreqs is an N_src array
    //Outputs: re_part and im_part are N_fq arrays holding the computed visibility.
    int N_fq = *N_fq_p;
    int N_src = *N_src_p;
    float lmin = *lmin;
    float lmax = *lmax;
    float mmin = *mmin;
    float mmax = *mmax;
    float beamfqmin = *beamfqmin;
    float beamfqmax = *beamfqmax;
    float coeff, dot=0, fq;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int sid = blockIdx.y * blockDim.y + threadIdx.y;

    if (tid >= N_fq || sid >= N_src) return;
    fq = freqs[tid];
    // Find the position in normalized texture array coordinates
    float l_norm = (src_dir[3*sid] - lmin)/(lmax - lmin);
    float m_norm = (src_dir[3*sid] - mmin)/(mmax - mmin);
    float fq_norm = (fq - beamfqmin)/(beamfqmax - beamfqmin);
    for (int i=0 ; i < 3 ; i++) {//compute the dot product of baseline and source direction
           dot += src_dir[3*sid+i] * baseline[i];
    }
    dot *= -2 * CL_M_PI_F;
    coeff = src_int[sid] * powf(fq/mfreqs[sid], src_index[sid]) * tex3D(tex, l_norm, m_norm, fq_norm);
    vis_arr[2*(N_src*tid + sid)  ] = coeff * cosf(fq*dot);
    vis_arr[2*(N_src*tid + sid)+1] = coeff * sinf(fq*dot);
}

__global__ void sum_vis(float *vis_arr, float *sum_vis_arr, int *N_fq_p, int *N_src_p) {
    int N_fq = *N_fq_p;
    int N_src = *N_src_p;
    int tid = blockIdx.x;
    if (tid >= N_fq) return;
    sum_vis_arr[2*tid  ] = 0;
    sum_vis_arr[2*tid+1] = 0;
    for(int i=0 ; i < N_src ; i++){//iterate over all sources
        sum_vis_arr[2*tid  ] += vis_arr[2*(N_src*tid+i)  ];
        sum_vis_arr[2*tid+1] += vis_arr[2*(N_src*tid+i)+1];
    }
}

int vis_sim(float *baseline, float *src_dir, float *src_int, float *src_index,
            float *freqs, float *mfreqs, float *vis_arr, float *beam_arr,
            int l, int m, int N_beam_fq, float lmin, float lmax, float mmin, float mmax,
            float beamfqmin, float beamfqmax, int N_fq, int N_src){
	float *dev_baseline, *dev_src_dir, *dev_src_int, *dev_src_index, *dev_freqs, *dev_mfreqs,
          *dev_vis_arr, *dev_sum_vis_arr, *dev_beam_arr, *dev_lmin, *dev_lmax,
          *dev_mmin, *dev_mmax, *dev_beamfqmin, *dev_beamfqmax;
    int *dev_N_fq, *dev_N_src;

    hipExtent beam_arr_size = make_hipExtent(l, m, N_beam_fq);
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
	// Allocate memory on the GPU, do we need to check for success on hipMalloc?
	HANDLE_ERROR(hipMalloc((void**) &dev_baseline,      3*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**) &dev_src_dir,       3*N_src*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**) &dev_src_int,       N_src*sizeof(float)));
    HANDLE_ERROR(hipMalloc((void**) &dev_src_index,     N_src*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**) &dev_freqs,         N_fq*sizeof(float)));
    HANDLE_ERROR(hipMalloc((void**) &dev_mfreqs,        N_src*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**) &dev_vis_arr,       2 * N_fq * N_src * sizeof(float)));	    
	HANDLE_ERROR(hipMalloc((void**) &dev_sum_vis_arr,   2 * N_fq * sizeof(float)));	    
    HANDLE_ERROR(hipMalloc((void**) &dev_N_fq,          sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**) &dev_N_src,         sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**) &dev_lmin,          sizeof(float)));
    HANDLE_ERROR(hipMalloc((void**) &dev_lmax,          sizeof(float)));
    HANDLE_ERROR(hipMalloc((void**) &dev_mmin,          sizeof(float)));
    HANDLE_ERROR(hipMalloc((void**) &dev_mmax,          sizeof(float)));
    HANDLE_ERROR(hipMalloc((void**) &dev_beamfqmin,     sizeof(float)));
    HANDLE_ERROR(hipMalloc((void**) &dev_beamfqmax,     sizeof(float)));
    
    //Allocate memory for beam_arr.  
    HANDLE_ERROR(hipMalloc3DArray((void**) &dev_beam_arr, &channelDesc, beam_arr_size));
	
	// Move the arrays onto the GPU
    hipMemcpy(dev_baseline,  baseline,  3*sizeof(float),         
                hipMemcpyHostToDevice);
    hipMemcpy(dev_src_dir,   src_dir,   3*N_src*sizeof(float),         
                hipMemcpyHostToDevice);    
    hipMemcpy(dev_src_int,   src_int,   N_src*sizeof(float),         
                hipMemcpyHostToDevice);    
    hipMemcpy(dev_src_index, src_index, N_src*sizeof(float),
                hipMemcpyHostToDevice);
    hipMemcpy(dev_freqs,     freqs,     N_fq*sizeof(float),         
                hipMemcpyHostToDevice);
    hipMemcpy(dev_mfreqs,    mfreqs,    N_src*sizeof(float),
                hipMemcpyHostToDevice);
    hipMemcpy(dev_N_fq,      &N_fq,      sizeof(int),    hipMemcpyHostToDevice);
    hipMemcpy(dev_N_src,     &N_src,     sizeof(int),    hipMemcpyHostToDevice);
    hipMemcpy(dev_lmin,      &lmin,      sizeof(float),  hipMemcpyHostToDevice);
    hipMemcpy(dev_lmax,      &lmax,      sizeof(float),  hipMemcpyHostToDevice);
    hipMemcpy(dev_mmin,      &mmin,      sizeof(float),  hipMemcpyHostToDevice);
    hipMemcpy(dev_mmax,      &mmax,      sizeof(float),  hipMemcpyHostToDevice);
    hipMemcpy(dev_beamfqmin, &beamfqmin, sizeof(float),  hipMemcpyHostToDevice);
    hipMemcpy(dev_beamfqmax, &beamfqmax, sizeof(float),  hipMemcpyHostToDevice);

    //Copy the beam_arr array onto the GPU
    cuda Memcpy3DParams copyParams = {0};
    copyParams.srcPtr   = make_hipPitchedPtr((void*)beam_arr,  beam_arr_size.width*sizeof(float), beam_arr_size.width, beam_arr_size.height);
    copyParams.dstArray = dev_beam_arr;
    copyParams.extent   = beam_arr_size;
    copyParams.kind     = cudaMemcpyHOstToDevice;
    hipMemcpy3D(&copyParams);

    //set Texture parameters
    tex.normalized = true;
    tex.filtermode = hipFilterModeLinear;
    tex.addressMode[0] = hipAddressModeBorder;
    tex.addressMode[1] = hipAddressModeBorder;
    tex.addressMode[2] = hipAddressModeBorder;
    

    //bind array to texture
    HANDLE_ERROR(hipBindTextureToArray(tex, dev_beam_arr, channelDesc));

    dim3 grid(N_fq, N_src);
    
	find_vis<<<grid,1>>>(dev_baseline, dev_src_dir, dev_src_int, dev_src_index, dev_freqs, dev_mfreqs, 
                        dev_N_fq, dev_N_src, dev_vis_arr, dev_beam_arr, dev_lmin, dev_lmax, 
                        dev_mmin, dev_mmax, dev_beamfqmin, dev_beamfqmax);
	sum_vis<<<N_fq,1>>>(dev_vis_arr, dev_sum_vis_arr, dev_N_fq, dev_N_src);
	
	// copy the array back
	hipMemcpy(vis_arr, dev_sum_vis_arr, 2 * N_fq * sizeof(float), hipMemcpyDeviceToHost);
	
    //frees memory allocated on GPU
    hipFree(dev_baseline);
    hipFree(dev_src_dir);
    hipFree(dev_src_int);
    hipFree(dev_src_index);
    hipFree(dev_freqs);
    hipFree(dev_mfreqs);
    hipFree(dev_vis_arr);
    hipFree(dev_sum_vis_arr);
    hipFree(dev_N_fq);
    hipFree(dev_N_src);
    hipFree(dev_beam_arr);
    hipFree(dev_lmin);
    hipFree(dev_lmax);
    hipFree(dev_mmin);
    hipFree(dev_mmax);
    hipFree(dev_beamfqmin);
    hipFree(dev_beamfqmax);

	return 0;
}
