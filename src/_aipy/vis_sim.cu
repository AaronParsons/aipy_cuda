#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <math.h>
#include "cuda_add.h"


__global__ void find_vis( float *baseline, float *src_dir, float *src_int, float *freqs, int N_fq, int N_src, float *re_part, float *im_part) {
	int tid = blockIdx.x;
    int c = 3*10**8;
    if (tid < N_fq){
        fq = freqs[tid];
        for(int i =0;i<N_src;i++){//iterate over all sources
            float dot = 0;
            for (int j = 0;j<3;j++){//compute the dot product of baseline and source direction
                   dot += src_dir[i][j]*baseline[j];
                }
            coeff = src_int[i]*(fq/mfreq)**src_index[i];
            re_part = coeff*cos(-2*pi*fq*dot/c);
            im_part = coeff*sin(-2*pi*fq*dot/c);
            }
    }
}

int *cuda_add(float *baseline, float *src_dir, float *src_int, 
            float *freqs, float *vis_arr,
            int N_fq, int N_src){
    float *re_part, *im_part;
	float *dev_baseline, *dev_src_dir, *dev_src_int,*dev_freqs,
          *dev_re_part, *dev_im_part;
    int dev_N_fq, dev_N_src;

	// Allocate memory on the GPU
	hipMalloc((void**) &dev_baseline,  3*sizeof(float));
	hipMalloc((void**) &dev_src_dir,   3*N_src*sizeof(float));
	hipMalloc((void**) &dev_src_int,   N_src*sizeof(float));
	hipMalloc((void**) &dev_src_freqs, N_fq*sizeof(float));
	hipMalloc((void**) &dev_re_part,   N_fq*sizeof(float));
	hipMalloc((void**) &dev_im_part,   N_fq*sizeof(float));	    
    hipMalloc((void**) &dev_N_fq,      sizeof(int));
	hipMalloc((void**) &dev_N_src,     sizeof(int));
	
	// Move the arrays onto the GPU
    hipMemcpy(dev_baseline,  baseline,  3*sizeof(float),         
                hipMemcpyHostToDevice);
    hipMemcpy(dev_src_dir,   src_dir,   3*N_src*sizeof(float),         
                hipMemcpyHostToDevice);    
    hipMemcpy(dev_src_int,   src_int,   N_src*sizeof(float),         
                hipMemcpyHostToDevice);    
    hipMemcpy(dev_src_freqs, src_freqs, N_fq*sizeof(float),         
                hipMemcpyHostToDevice);
    hipMemcpy(dev_re_part,   re_part,   N_fq*sizeof(float),         
                hipMemcpyHostToDevice);
    hipMemcpy(dev_im_part,   im_part,   N_fq*sizeof(float),         
                hipMemcpyHostToDevice);
    hipMemcpy(dev_N_fq,      N_fq,      sizeof(int),  hipMemcpyHostToDevice);
    hipMemcpy(dev_N_src,     N_src,     sizeof(int)), hipMemcpyHostToDevice);


	find_vis<<<N_fq,1>>>(baseline, src_dir, src_int, freqs, 
                        N_fq, N_src, 
                        re_part, im_part);
	
	// copy the array back
	hipMemcpy(re_part, dev_re_part, N_fq * sizeof(float), 
                hipMemcpyDeviceToHost);
    hipMemcpy(im_part, dev_im_part, N_fq * sizeof(float),
                hipMemcpyDeviceToHost);
	
    //frees memory allocated on GPU
    
    hipFree(dev_baseline);
    hipFree(dev_src_dir);
    hipFree(dev_src_int);
    hipFree(dev_src_freqs);
    hipFree(dev_re_part);
    hipFree(dev_im_part);
    hipFree(dev_N_fq);
    hipFree(dev_N_src);
    
    //interleave re_part and im_part in the output array
    for(int i = 0;i<N_fq;i++){
        *vis_arr[2*i] = re_part[i];
        *vis_arr[2*i+1] = im_part[i];
        }

	return 1;
}
/
