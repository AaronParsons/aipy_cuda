#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void add( int *a, int *b, int *c, int N) {
	int tid = blockIdx.x;
	if (tid < N)
		c[tid] = a[tid] +b[tid];
}

int cuda_add(int* a, int* b, int N){
	int *dev_a, *dev_b, *dev_c;
	int c[N];

	// Allocate memory on the GPU
	hipMalloc((void**) &dev_a, N*sizeof(int));
	hipMalloc((void**) &dev_b, N*sizeof(int));
	hipMalloc((void**) &dev_c, N*sizeof(int));
	
	// Move the arrays onto the GPU
	hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
				
	hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);
				
	add<<<N,1>>>(dev_a, dev_b, dev_c, N);
	
	// copy the array back
	hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);
	
	for (int i=0;i<N;i++){
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
		}
		
	return 1;
}

